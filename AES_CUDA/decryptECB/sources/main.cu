#include "hip/hip_runtime.h"
#include "AES.h"
#include <stdio.h>

__global__
void decryptECB(state_type in[IN_LEN], state_type out[OUT_LEN],
		w_type key[KEY_LEN]) {
    for (int i=0; i< IN_LEN;i++)
        printf("%02x ", (unsigned char) in[i]);

    printf("\n");
	w_type w[KEY_ROUND];
	keyExpansion(key, w);
	invCipher(in, out, w);
}

int main() {

	uint8_t key[] = { 0x60, 0x3d, 0xeb, 0x10, 0x15, 0xca, 0x71, 0xbe, 0x2b,
			0x73, 0xae, 0xf0, 0x85, 0x7d, 0x77, 0x81, 0x1f, 0x35, 0x2c, 0x07,
			0x3b, 0x61, 0x08, 0xd7, 0x2d, 0x98, 0x10, 0xa3, 0x09, 0x14, 0xdf,
			0xf4 };
	uint8_t right[] = { 0xf3, 0xee, 0xd1, 0xbd, 0xb5, 0xd2, 0xa0, 0x3c, 0x06,
			0x4b, 0x5a, 0x7e, 0x3d, 0xb1, 0x81, 0xf8 };

	uint8_t plain[] = { 0x6b, 0xc1, 0xbe, 0xe2, 0x2e, 0x40, 0x9f, 0x96, 0xe9,
			0x3d, 0x7e, 0x11, 0x73, 0x93, 0x17, 0x2a };

	state_type out[OUT_LEN];

    uint8_t *key_gpu, *right_gpu, *out_gpu;
    float elapsedTime;
    hipEvent_t start, stop; // pomiar czasu wykonania j?dra

    checkCudaErrors(hipSetDevice(0));

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMalloc(&key_gpu, sizeof(uint8_t)*32));
    checkCudaErrors(hipMalloc(&out_gpu, sizeof(uint8_t)*(16)));
    checkCudaErrors(hipMalloc(&right_gpu, sizeof(uint8_t)*(16)));

    checkCudaErrors(hipMemcpy(key_gpu, key, sizeof(uint8_t)*32, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(right_gpu, right, sizeof(uint8_t)*(16), hipMemcpyHostToDevice));

	decryptECB<<<1, 1>>>(right_gpu, out_gpu, key_gpu);

    checkCudaErrors(hipGetLastError());
    
    checkCudaErrors(hipMemcpy(out, out_gpu, sizeof(uint8_t)*(16), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    printf("GPU (kernel) time = %.3f ms\n", elapsedTime);

    checkCudaErrors(hipFree(key_gpu));
    checkCudaErrors(hipFree(out_gpu));
    checkCudaErrors(hipFree(right_gpu));

    checkCudaErrors(hipDeviceReset());

	for (int i = 0; i < OUT_LEN; i++) {
		printf("%02x ", (unsigned char) out[i]);
	}

	if (0 == memcmp((char*) out, (char*) plain, 16)) {
		printf("SUCCESS!\n");
		return (0);
	} else {
		printf("FAILURE!\n");
		return (1);
	}
}
