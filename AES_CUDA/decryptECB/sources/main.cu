#include "hip/hip_runtime.h"
#include "AES.h"
#include <stdio.h>

__global__
void decryptECB(state_type in[IN_LEN], state_type out[OUT_LEN],
		w_type key[KEY_LEN], int size) {
	w_type w[KEY_ROUND];
	keyExpansion(key, w);
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    if (idx < size)
    {

	    invCipher(in+idx*16, out+idx*16, w);
    }
}

int main() {
    std::ifstream in_file;
	in_file.open("/home/silver/My-projects/CUDA/samples/0_Simple/aes_project/AES_CUDA/decryptECB/data/encrypted.txt", std::ios::binary);
    std::size_t file_size = std::experimental::filesystem::file_size("/home/silver/My-projects/CUDA/samples/0_Simple/aes_project/AES_CUDA/decryptECB/data/encrypted.txt");
    // int padding = 16-(file_size % 16);
    // std::cout<<"padding is "<<padding<<"\n";
    char encrypted[file_size];
    int N = file_size/16;

	std::string text;
	if(!in_file.is_open())
	{
		std::cout<<"file not open\n";
	}

    std::cout<<"file size: "<<file_size<<"\n";
    in_file.read(encrypted, file_size);
	in_file.close();

    // for(int i = 0; i<padding;i++)
    // {
    //     encrypted[file_size+i] = ' ';
    // }
    // uint8_t encrypted[] = {140, 99, 39, 200, 100, 130, 179, 140, 106, 210, 92, 170, 150, 241, 255, 105};

	uint8_t key[] = { 0x60, 0x3d, 0xeb, 0x10, 0x15, 0xca, 0x71, 0xbe, 0x2b,
			0x73, 0xae, 0xf0, 0x85, 0x7d, 0x77, 0x81, 0x1f, 0x35, 0x2c, 0x07,
			0x3b, 0x61, 0x08, 0xd7, 0x2d, 0x98, 0x10, 0xa3, 0x09, 0x14, 0xdf,
			0xf4 };

	state_type out[file_size];

    uint8_t *key_gpu, *encrypted_gpu, *out_gpu;
    float elapsedTime;
    hipEvent_t start, stop; // pomiar czasu wykonania j?dra

    // int pom;
    for(int i=0;i<file_size;i++)
        
        std::cout<<(int)encrypted[i]<<" ";
    std::cout<<"\n";

    checkCudaErrors(hipSetDevice(0));

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipMalloc(&key_gpu, sizeof(uint8_t)*32));
    checkCudaErrors(hipMalloc(&out_gpu, sizeof(uint8_t)*(file_size)));
    checkCudaErrors(hipMalloc(&encrypted_gpu, sizeof(uint8_t)*(file_size)));

    checkCudaErrors(hipMemcpy(key_gpu, key, sizeof(uint8_t)*32, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(encrypted_gpu, encrypted, sizeof(uint8_t)*(file_size), hipMemcpyHostToDevice));

	decryptECB<<<(N+63)/64, 64>>>(encrypted_gpu, out_gpu, key_gpu, N);

    checkCudaErrors(hipGetLastError());
    
    checkCudaErrors(hipMemcpy(out, out_gpu, sizeof(uint8_t)*(file_size), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(stop, 0));

    for(int i=0;i<file_size;i++)
    {

        std::cout<<(char)out[i]<<" ";
        // itoa(out[i], encrypted[i], 10);
    }

    std::cout<<"\n";

    int padding = (int)out[file_size-1];
    std::cout<<"padding is "<<padding<<"\n";
    std::ofstream out_file;
    out_file.open("/home/silver/My-projects/CUDA/samples/0_Simple/aes_project/AES_CUDA/decryptECB/data/plain.txt", std::ios::binary);
    out_file.write((char *)out, file_size-padding);
    out_file.close();
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    printf("GPU (kernel) time = %.3f ms\n", elapsedTime);

    checkCudaErrors(hipFree(key_gpu));
    checkCudaErrors(hipFree(out_gpu));
    checkCudaErrors(hipFree(encrypted_gpu));

    checkCudaErrors(hipDeviceReset());
}
