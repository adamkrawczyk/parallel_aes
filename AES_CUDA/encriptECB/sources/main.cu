#include "hip/hip_runtime.h"
#include "AES.h"

__global__
static void ecb_encrypt_kernel(state_type* in, state_type* out, w_type* key, int size)
{
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    if (idx < size){
        state_type state[Nb][Nb];

        // arrayTransformOneDim(in+idx*16, state);
        #pragma unroll
        for (int i = 0; i < Nb; i++) {
            #pragma unroll
            for (int j = 0; j < Nb; j++) {
                state[i][j] = in[j + Nb * i + idx*16];
            }
        }
        
        
        // addRoundKey(state, key);
        #pragma unroll
        for (int j = 0; j < Nb; j++) {
            #pragma unroll
            for (int i = 0; i < Nb; i++) {
                state[j][i] = state[j][i] ^ key[i + Nb * j]; //"w" pseudo conversion to a 2-dimensional array
            }

        }


        #pragma unroll
        for (int round = 1; round <= Nr - 1; round++) {
            // subBytes(state);
            #pragma unroll
            for (int i = 0; i < Nb; i++) {
                #pragma unroll
                for (int j = 0; j < Nb; j++) {
                    state[j][i] = sbox[state[j][i]];
                }
            }
            // shiftRows(state);
            #pragma unroll
            for (int numberOfShifts = 0; numberOfShifts < Nb; numberOfShifts++) {
                #pragma unroll
                for (int j = 0; j < numberOfShifts; j++) {
                    state_type tmp = state[0][numberOfShifts];
                    #pragma unroll
                    for (int i = 0; i < Nb - 1; i++) {
                        state[i][numberOfShifts] = state[i + 1][numberOfShifts];
                    }
                    state[Nb - 1][numberOfShifts] = tmp;
                }
            }
            // mixColumns(state);
            state_type r[Nb];
            state_type a[Nb];
            state_type b[Nb];

            #pragma unroll
            for (int i = 0; i < Nb; i++) {
                #pragma unroll
                for(int j = 0; j < Nb; j++)
                {
                    r[j] = state[i][j];
                }

                //Rijndael_MixColumns https://en.wikipedia.org/wiki/Rijndael_MixColumns
                #pragma unroll
                for (int c = 0; c < 4; c++) {
                    a[c] = r[c];
                    b[c] = (r[c] << 1) ^ (0x1B * (1 & (r[c] >> 7)));
                }

                r[0] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
                r[1] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
                r[2] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
                r[3] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0];

                #pragma unroll
                for(int j = 0; j < Nb; j++)
                {
                    state[i][j] = r[j];
                }

            }
            // addRoundKey(state, (key + round * Nb * Nb));
            #pragma unroll
            for (int j = 0; j < Nb; j++) {
                #pragma unroll
                for (int i = 0; i < Nb; i++) {
                    state[j][i] = state[j][i] ^ key[i + Nb * j + round*Nb*Nb]; //"w" pseudo conversion to a 2-dimensional array
                }
        
            }
        }

        // subBytes(state);
        #pragma unroll
        for (int i = 0; i < Nb; i++) {
            #pragma unroll
            for (int j = 0; j < Nb; j++) {
                state[j][i] = sbox[state[j][i]];
            }
        }
        // shiftRows(state);
        #pragma unroll
        for (int numberOfShifts = 0; numberOfShifts < Nb; numberOfShifts++) {
            #pragma unroll
            for (int j = 0; j < numberOfShifts; j++) {
                state_type tmp = state[0][numberOfShifts];
                #pragma unroll
                for (int i = 0; i < Nb - 1; i++) {
                    state[i][numberOfShifts] = state[i + 1][numberOfShifts];
                }
                state[Nb - 1][numberOfShifts] = tmp;
            }
        }
        // addRoundKey(state, (key + Nr * Nb * Nb));
        #pragma unroll
        for (int j = 0; j < Nb; j++) {
            #pragma unroll
            for (int i = 0; i < Nb; i++) {
                state[j][i] = state[j][i] ^ key[i + Nb * j+Nr * Nb * Nb]; //"w" pseudo conversion to a 2-dimensional array
            }
    
        }

        // arrayTransformTwoDim(out+idx*16, state);
        #pragma unroll
        for (int i = 0; i < Nb; i++) {
            #pragma unroll
            for (int j = 0; j < Nb; j++) {
                out[j + Nb * i+idx*16] = state[i][j];
            }
        }
        // encriptECB(in+idx*16, out+idx*16, key);
    }
}

int main() {
	std::ifstream in_file;
    char in_file_path[] = "/home/silver/My-projects/CUDA/samples/0_Simple/aes_project/AES_CUDA/encriptECB/data/sw.png";
	in_file.open(in_file_path, std::ios::binary);
    std::size_t file_size = std::experimental::filesystem::file_size(in_file_path);
    int padding = 16 - (file_size % 16) + 16;
    char plain[file_size+padding];
    int N = (file_size+padding)/16;

	std::string text;
	if(!in_file.is_open())
	{
		std::cout<<"file not open\n";
	}

    in_file.read(plain, file_size);
	in_file.close();

    // fill padding with zeros
    for(int i=0;i<padding-1;i++)
    {
        plain[file_size+i] = 0;
    }
    plain[file_size+padding-1] = padding;

	uint8_t key[] = { 0x60, 0x3d, 0xeb, 0x10, 0x15, 0xca, 0x71, 0xbe, 0x2b,
        0x73, 0xae, 0xf0, 0x85, 0x7d, 0x77, 0x81, 0x1f, 0x35, 0x2c, 0x07,
        0x3b, 0x61, 0x08, 0xd7, 0x2d, 0x98, 0x10, 0xa3, 0x09, 0x14, 0xdf,
        0xf4 };


	state_type out[file_size+padding];

    uint8_t *key_gpu, *plain_gpu, *out_gpu;
    float elapsedTime, kernelTime;
    hipEvent_t start, kernel_start, kernel_stop, stop; // pomiar czasu wykonania j?dra

    checkCudaErrors(hipSetDevice(0));

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&kernel_start));
    checkCudaErrors(hipEventCreate(&kernel_stop));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
    w_type w[KEY_ROUND];
    keyExpansion(key, w);

    checkCudaErrors(hipMalloc(&key_gpu, sizeof(uint8_t)*KEY_ROUND));
    checkCudaErrors(hipMalloc(&out_gpu, sizeof(uint8_t)*(file_size+padding)));
    checkCudaErrors(hipMalloc(&plain_gpu, sizeof(uint8_t)*(file_size+padding)));

    checkCudaErrors(hipMemcpy(key_gpu, w, sizeof(uint8_t)*KEY_ROUND, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(plain_gpu, plain, sizeof(uint8_t)*(file_size+padding), hipMemcpyHostToDevice));

    checkCudaErrors(hipEventRecord(kernel_start, 0));

    ecb_encrypt_kernel<<<(N+255)/256, 256>>>(plain_gpu, out_gpu, key_gpu, N);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(kernel_stop, 0));

    checkCudaErrors(hipMemcpy(out, out_gpu, sizeof(uint8_t)*(file_size+padding), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(stop, 0));

	std::ofstream out_file;
    out_file.open("/home/silver/My-projects/CUDA/samples/0_Simple/aes_project/AES_CUDA/encriptECB/data/encrypted.txt", std::ios::binary);

    std::cout<<"\n"<<"full file size is "<<file_size+padding<<"\n";
    out_file.write((char *)out, file_size+padding);
    out_file.close();

    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaErrors(hipEventElapsedTime(&kernelTime, kernel_start, kernel_stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    printf("GPU time = %.3f ms\n", elapsedTime);
    printf("GPU kernel time = %.3f ms\n", kernelTime);
    float throughput = 1000/kernelTime*N*16.0f*8.0f/(1024.0f*1024.0f*1024.0f);
    printf("Throughput = %.3f Gb/s\n", throughput);


    checkCudaErrors(hipFree(key_gpu));
    checkCudaErrors(hipFree(plain_gpu));

    checkCudaErrors(hipDeviceReset());
    return (0);
}